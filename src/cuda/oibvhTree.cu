#include "hip/hip_runtime.h"
#include <oibvh/cuda/oibvh.cuh>
#include <oibvh/cuda/oibvhTree.hpp>

#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include <fstream>

OibvhTree::OibvhTree(const std::shared_ptr<Mesh> mesh) : m_mesh(mesh), m_buildDone(false)
{
    deviceMalloc(&m_devicePositions, 10000000);
    deviceMalloc(&m_deviceFaces, 10000000);
    deviceMalloc(&m_deviceAabbs, 10000000);
    setup();
}

OibvhTree::OibvhTree(const std::shared_ptr<OibvhTree> other, const std::shared_ptr<Mesh> mesh)
    : m_mesh(mesh)
    , m_buildDone(other->m_buildDone)
    , m_aabbTree(other->m_aabbTree)
    , m_faces(other->m_faces)
    , m_positions(other->m_positions)
    , m_scheduleParams(other->m_scheduleParams)
{
    deviceMalloc(&m_devicePositions, 10000000);
    deviceMalloc(&m_deviceFaces, 10000000);
    deviceMalloc(&m_deviceAabbs, 10000000);

    setup();
}

OibvhTree::~OibvhTree()
{
    hipFree(m_devicePositions);
    hipFree(m_deviceFaces);
    hipFree(m_deviceAabbs);
}

unsigned int OibvhTree::getDepth() const
{
    return ilog2(m_aabbTree.size());
}

unsigned int OibvhTree::getPrimCount() const
{
    return m_faces.size();
}

void OibvhTree::schedulingParameters(const unsigned int entryLevel,
                                     const unsigned int realCount,
                                     const unsigned int threadsPerGroup)
{
    m_scheduleParams.clear();

    unsigned int l = entryLevel;
    unsigned int r = realCount;
    unsigned int g = std::min(threadsPerGroup, next_power_of_two(r));
    unsigned int t = (r + g - 1) / g * g;

    unsigned int rLast, gLast, tLast;
    while (1)
    {
        rLast = r;
        tLast = t;
        gLast = g;
        m_scheduleParams.push_back({l, rLast, tLast, gLast});

        if (l >= ilog2(gLast) + 1)
            l = l - ilog2(gLast) - 1;
        else
            break;

        r = tLast / gLast;
        r = (r + 1) / 2;
        g = std::min(gLast, next_power_of_two(r));
        t = (r + g - 1) / g * g;
    }
}

void OibvhTree::setup()
{
    if (!m_buildDone)
    {
        std::cout << "---Set up oibvh tree---" << std::endl;
        for (int i = 0; i < m_mesh->m_facesCount; i++)
        {
            m_faces.push_back(
                glm::uvec3(m_mesh->m_indices[i * 3], m_mesh->m_indices[i * 3 + 1], m_mesh->m_indices[i * 3 + 2]));
        }
        for (auto vertex : m_mesh->m_vertices)
        {
            m_positions.push_back(vertex.m_position);
        }
        std::cout << "faces count: " << m_faces.size() << std::endl;
        std::cout << "vertices count: " << m_positions.size() << std::endl;
        std::cout << std::endl;
    }
}

void OibvhTree::refit()
{
    // std::cout << "---Refit---" << std::endl;
    for (int i = 0; i < m_mesh->m_verticesCount; i++)
    {
        m_positions[i] = m_mesh->m_vertices[i].m_position;
    }
    float elapsed_ms = 0.0f;
    const unsigned int primitive_count = m_faces.size();
    const unsigned int vertex_count = m_positions.size();
    const unsigned int oibvh_size = oibvh_get_size(primitive_count);
    const unsigned int oibvh_internal_node_count = oibvh_size - primitive_count;
    glm::vec3* d_positions = m_devicePositions;
    glm::uvec3* d_faces = m_deviceFaces;
    aabb_box_t* d_aabbs = m_deviceAabbs;
    deviceMemcpy(d_positions, m_positions.data(), vertex_count);
    deviceMemcpy(d_faces, m_faces.data(), primitive_count);

    elapsed_ms = kernelLaunch([&]() {
        dim3 blockSize = dim3(256);
        int bx = (primitive_count + blockSize.x - 1) / blockSize.x;
        dim3 gridSize = dim3(bx);
        calculate_aabb_kernel<<<gridSize, blockSize>>>(
            d_faces, d_positions, primitive_count, d_aabbs + oibvh_internal_node_count);
    });
    // std::cout << "Refit: AABBs calculation took: " << elapsed_ms << "ms" << std::endl;

    for (int k = 0; k < m_scheduleParams.size(); k++)
    {
        elapsed_ms = kernelLaunch([&]() {
            dim3 blockSize = dim3(m_scheduleParams[k].m_threadsPerGroup);
            dim3 gridSize = dim3(m_scheduleParams[k].m_threads / m_scheduleParams[k].m_threadsPerGroup);
            oibvh_tree_construction_kernel<<<gridSize, blockSize>>>(m_scheduleParams[k].m_entryLevel,
                                                                    m_scheduleParams[k].m_realCount,
                                                                    primitive_count,
                                                                    m_scheduleParams[k].m_threadsPerGroup,
                                                                    d_aabbs);
        });
        // std::cout << "  oibvh refit construct kernel took: " << elapsed_ms << "ms" << std::endl;
    }

    hostMemcpy(m_aabbTree.data(), d_aabbs, oibvh_size);
}

void OibvhTree::build()
{
    std::cout << "---Build oibvh tree---" << std::endl;
    int dev;
    float elapsed_ms = 0.0f;
    hipGetDevice(&dev);
    std::cout << "device id: " << dev << std::endl;
    const unsigned int primitive_count = m_faces.size();
    const unsigned int vertex_count = m_positions.size();
    const unsigned int oibvh_size = oibvh_get_size(primitive_count);

    const unsigned int oibvh_internal_node_count = oibvh_size - primitive_count;
    glm::vec3* d_positions;
    glm::uvec3* d_faces;
    aabb_box_t* d_aabbs;
    unsigned int* d_mortons;
    deviceMalloc(&d_positions, vertex_count);
    deviceMalloc(&d_faces, primitive_count);
    deviceMalloc(&d_aabbs, oibvh_size);
    deviceMalloc(&d_mortons, primitive_count);
    deviceMemcpy(d_positions, m_positions.data(), vertex_count);
    deviceMemcpy(d_faces, m_faces.data(), primitive_count);

    elapsed_ms = kernelLaunch([&]() {
        dim3 blockSize = dim3(256);
        int bx = (primitive_count + blockSize.x - 1) / blockSize.x;
        dim3 gridSize = dim3(bx);
        calculate_aabb_and_morton_kernel<<<gridSize, blockSize>>>(
            d_faces, d_positions, primitive_count, m_mesh->m_aabb, d_aabbs + oibvh_internal_node_count, d_mortons);
    });
    std::cout << "AABBs and mortons calculation took: " << elapsed_ms << "ms" << std::endl;

#if 0
     // check result
     aabb_box_t* temp_aabbs;
     hostMalloc(&temp_aabbs, primitive_count);
     hostMemcpy(temp_aabbs, d_aabbs + oibvh_internal_node_count, primitive_count);
     aabb_box_t aabb;
     std::cout << sizeof(aabb_box_t) << std::endl;
     aabb.minimum = glm::vec3(1e10);
     aabb.maximum = glm::vec3(-1e10);
     for (int i = 0; i < primitive_count; i++)
    {
        aabb.maximum = glm::max(aabb.maximum, temp_aabbs[i].maximum);
        aabb.minimum = glm::min(aabb.minimum, temp_aabbs[i].minimum);
    }
     aabb == m_mesh->m_aabb ? std::cout << "aabb is correct" << std::endl : std::cout << "aabb is wrong" << std::endl;
     delete[] temp_aabbs;
#endif

    unsigned int* d_mortons_copy;
    deviceMalloc(&d_mortons_copy, primitive_count);
    hipMemcpy(d_mortons_copy, d_mortons, primitive_count * sizeof(unsigned int), hipMemcpyDeviceToDevice);
    thrust::device_ptr<unsigned int> d_mortons_copy_ptr(d_mortons_copy);
    thrust::device_ptr<unsigned int> d_mortons_ptr(d_mortons);
    thrust::device_ptr<glm::uvec3> d_faces_ptr(d_faces);
    thrust::device_ptr<aabb_box_t> d_aabbs_leaf_ptr(d_aabbs + oibvh_internal_node_count);
    elapsed_ms = kernelLaunch([&]() {
        thrust::stable_sort_by_key(d_mortons_ptr, d_mortons_ptr + primitive_count, d_faces_ptr);
        thrust::stable_sort_by_key(d_mortons_copy_ptr, d_mortons_copy_ptr + primitive_count, d_aabbs_leaf_ptr);
    });
    std::cout << "Sorting took: " << elapsed_ms << "ms" << std::endl;
    hipFree(d_mortons_copy);

#if 0
    // print result
    aabb_box_t* temp_aabbs;
    hostMalloc(&temp_aabbs, primitive_count);
    hostMemcpy(temp_aabbs, d_aabbs + oibvh_internal_node_count, primitive_count);
    glm::uvec3* temp_faces;
    hostMalloc(&temp_faces, primitive_count);
    hostMemcpy(temp_faces, d_faces, primitive_count);
    for (int i = 0; i < 100; i++)
    {
        std::cout << temp_aabbs[i].minimum << "," << temp_aabbs[i].maximum << std::endl;
        std::cout << m_positions[temp_faces[i].x] << "," << m_positions[temp_faces[i].y] << "," << m_positions[temp_faces[i].z]
                  << std::endl;
    }
#endif

    const unsigned int primitiveCountNextPower2 = next_power_of_two(primitive_count);
    const unsigned int tHeight = ilog2(primitiveCountNextPower2) + 1;
    const unsigned int tLeafLev = tHeight - 1;
    unsigned int entryLevel = tLeafLev - 1;
    const unsigned int virtualLeafCount = primitiveCountNextPower2 - primitive_count;
    unsigned int entryLevelSize = oibvh_level_real_node_count(entryLevel, tLeafLev, virtualLeafCount);

    schedulingParameters(entryLevel, entryLevelSize, THREADS_PER_BLOCK);

#if 0
    // print result
    std::cout << "m_scheduleParams: " << std::endl;
    for (auto param : m_scheduleParams)
    {
        std::cout << param.m_entryLevel << "," << param.m_realCount << "," << param.m_threadsPerGroup << ","
                  << param.m_threads << std::endl;
    }
#endif

    std::cout << "kernel count: " << m_scheduleParams.size() << std::endl;

    for (int k = 0; k < m_scheduleParams.size(); k++)
    {
        std::cout << "kernel" << k << std::endl;
        std::cout << "  entry level: " << m_scheduleParams[k].m_entryLevel << std::endl;
        std::cout << "  real nodes: " << m_scheduleParams[k].m_realCount << std::endl;
        std::cout << "  total threads: " << m_scheduleParams[k].m_threads << std::endl;
        std::cout << "  group size: " << m_scheduleParams[k].m_threadsPerGroup << std::endl;
        std::cout << "  group count: " << m_scheduleParams[k].m_threads / m_scheduleParams[k].m_threadsPerGroup
                  << std::endl;
        elapsed_ms = kernelLaunch([&]() {
            dim3 blockSize = dim3(m_scheduleParams[k].m_threadsPerGroup);
            dim3 gridSize = dim3(m_scheduleParams[k].m_threads / m_scheduleParams[k].m_threadsPerGroup);
            oibvh_tree_construction_kernel2<<<gridSize, blockSize>>>(m_scheduleParams[k].m_entryLevel,
                                                                     m_scheduleParams[k].m_realCount,
                                                                     primitive_count,
                                                                     m_scheduleParams[k].m_threadsPerGroup,
                                                                     d_aabbs);
        });
        std::cout << "  oibvh contruct kernel took: " << elapsed_ms << "ms" << std::endl;
    }
    std::cout << "count of node in oibvh tree: " << oibvh_size << std::endl;

#if 0
    // log result
    aabb_box_t* temp_aabbs;
    hostMalloc(&temp_aabbs, oibvh_size);
    hostMemcpy(temp_aabbs, d_aabbs, oibvh_size);
    std::ofstream outfile;
    outfile.open("..//..//..//logs//bvh_log.txt");
    for (int i = 0; i < oibvh_size; i++)
    {
        outfile << temp_aabbs[i] << std::endl;
    }
    // std::cout << m_mesh->m_aabb.minimum << "," << m_mesh->m_aabb.maximum << std::endl;
#endif

    // copy result to host
    m_aabbTree.resize(oibvh_size);
    hostMemcpy(m_aabbTree.data(), d_aabbs, oibvh_size);
    hostMemcpy(m_faces.data(), d_faces, primitive_count);

    // free memory
    hipFree(d_positions);
    hipFree(d_faces);
    hipFree(d_aabbs);
    hipFree(d_mortons);

    // build done
    m_buildDone = true;
    std::cout << std::endl;
}