#include "hip/hip_runtime.h"
#include <oibvh/cuda/collide.cuh>
#include <oibvh/cuda/oibvh.cuh>
#include <oibvh/cuda/utils.cuh>

#include <>

#define min(x, y) (x < y ? x : y)
#define LOCALMEMSIZE 2048

__device__ inline bool overlap(const aabb_box_t& aabb1, const aabb_box_t& aabb2)
{
    return (aabb1.m_minimum.x <= aabb2.m_maximum.x && aabb1.m_maximum.x >= aabb2.m_minimum.x) &&
        (aabb1.m_minimum.y <= aabb2.m_maximum.y && aabb1.m_maximum.y >= aabb2.m_minimum.y) &&
        (aabb1.m_minimum.z <= aabb2.m_maximum.z && aabb1.m_maximum.z >= aabb2.m_minimum.z);
}

__device__ inline void read_information(unsigned int* sharedAabbOffsets,
                                        unsigned int* sharedPrimOffsets,
                                        unsigned int* sharedPrimCount,
                                        unsigned int aabbIndex,
                                        unsigned int layoutLength,
                                        unsigned int& aabbOffset,
                                        unsigned int& primOffset,
                                        unsigned int& primitiveCount)
{
    int l = 0;
    int r = layoutLength - 1;
    int m;
    int idxLayout;
    while (l <= r)
    {
        m = (l + r) / 2;
        if (sharedAabbOffsets[m] <= aabbIndex)
        {
            l = m + 1;
            idxLayout = m;
        }
        else
            r = m - 1;
    }
    aabbOffset = sharedAabbOffsets[idxLayout];
    primOffset = sharedPrimOffsets[idxLayout];
    primitiveCount = sharedPrimCount[idxLayout];
}

__device__ inline void read_information(unsigned int* sharedPrimOffsets,
                                        unsigned int* sharedVertexOffsets,
                                        unsigned int primIndex,
                                        unsigned int layoutLength,
                                        unsigned int& bvhIndex,
                                        unsigned int& primOffset,
                                        unsigned int& vertexOffset)
{
    int l = 0;
    int r = layoutLength - 1;
    int m;
    int idxLayout;
    while (l <= r)
    {
        m = (l + r) / 2;
        if (sharedPrimOffsets[m] <= primIndex)
        {
            l = m + 1;
            idxLayout = m;
        }
        else
            r = m - 1;
    }
    bvhIndex = idxLayout;
    primOffset = sharedPrimOffsets[idxLayout];
    vertexOffset = sharedVertexOffsets[idxLayout];
}

__global__ void traversal_kernel(bvtt_node_t* src,
                                 bvtt_node_t* dst,
                                 aabb_box_t* aabbs,
                                 tri_pair_node_t* triPairs,
                                 unsigned int* aabbOffsets,
                                 unsigned int* primOffsets,
                                 unsigned int* primCounts,
                                 unsigned int* nextBvttSize,
                                 unsigned int* triPairCount,
                                 unsigned int layoutLength,
                                 unsigned int bvttSize,
                                 unsigned int expandLevels)
{
    unsigned int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int localIndex = threadIdx.x;
    __shared__ unsigned int sharedAabbOffsets[256];
    __shared__ unsigned int sharedPrimOffsets[256];
    __shared__ unsigned int sharedPrimCounts[256];
    if (localIndex < layoutLength)
    {

        sharedAabbOffsets[localIndex] = aabbOffsets[localIndex];
        sharedPrimOffsets[localIndex] = primOffsets[localIndex];
        sharedPrimCounts[localIndex] = primCounts[localIndex];
    }
    __syncthreads();

    unsigned int aabbOffsetA, aabbOffsetB;
    unsigned int startImplicitIdxA, startImplicitIdxB;
    unsigned int expandNodeCountA = 0;
    unsigned int expandNodeCountB = 0;
    unsigned int realIndexA, realIndexB;
    unsigned int virtualCountA, virtualCountB;
    unsigned int levelA, levelB;
    unsigned int leafLevA, leafLevB;
    unsigned int numData = 0;

    if (globalIndex < bvttSize)
    {
        bvtt_node_t node = src[globalIndex];
        const unsigned int aabbIndexA = node.m_aabbIndex[0];
        const unsigned int aabbIndexB = node.m_aabbIndex[1];
        aabb_box_t aabbA = aabbs[aabbIndexA];
        aabb_box_t aabbB = aabbs[aabbIndexB];

        if (overlap(aabbA, aabbB)) // AABB overlap
        {
            // expand bvtt node
            unsigned int primOffsetA, primitiveCountA; // A oibvh tree
            unsigned int primOffsetB, primitiveCountB; // B oibvh tree
            read_information(sharedAabbOffsets,
                             sharedPrimOffsets,
                             sharedPrimCounts,
                             aabbIndexA,
                             layoutLength,
                             aabbOffsetA,
                             primOffsetA,
                             primitiveCountA);
            read_information(sharedAabbOffsets,
                             sharedPrimOffsets,
                             sharedPrimCounts,
                             aabbIndexB,
                             layoutLength,
                             aabbOffsetB,
                             primOffsetB,
                             primitiveCountB);
            const unsigned int primCountNextPower2A = next_power_of_two(primitiveCountA);
            const unsigned int primCountNextPower2B = next_power_of_two(primitiveCountB);
            virtualCountA = primCountNextPower2A - primitiveCountA;
            virtualCountB = primCountNextPower2B - primitiveCountB;
            leafLevA = ilog2(primCountNextPower2A);
            leafLevB = ilog2(primCountNextPower2B);
            realIndexA = aabbIndexA - aabbOffsetA;
            realIndexB = aabbIndexB - aabbOffsetB;
            const unsigned int implicitIndexA = oibvh_real_to_implicit(realIndexA, leafLevA, virtualCountA);
            const unsigned int implicitIndexB = oibvh_real_to_implicit(realIndexB, leafLevB, virtualCountB);
            levelA = ilog2(implicitIndexA + 1);
            levelB = ilog2(implicitIndexB + 1);

            if (levelA == leafLevA && levelB == leafLevB) // a and b are both at leaf node
            {
                const unsigned int primIndexA = primOffsetA + implicitIndexA + 1 - (1 << leafLevA);
                const unsigned int primIndexB = primOffsetB + implicitIndexB + 1 - (1 << leafLevB);
                const tri_pair_node_t triPair{primIndexA, primIndexB};
                const unsigned int triPairIndex = atomicAdd(triPairCount, 1u);
                triPairs[triPairIndex] = triPair;
            }
            else
            {
                const unsigned int nextLevelA = min(leafLevA, levelA + expandLevels);
                const unsigned int nextLevelB = min(leafLevB, levelB + expandLevels);
                startImplicitIdxA = oibvh_get_most_left_descendant_implicitIdx(implicitIndexA, nextLevelA - levelA);
                startImplicitIdxB = oibvh_get_most_left_descendant_implicitIdx(implicitIndexB, nextLevelB - levelB);
                const unsigned int mostRightValidImplicitIdxA =
                    oibvh_get_most_right_valid_implicitIdx(nextLevelA, leafLevA, virtualCountA);
                const unsigned int mostRightValidImplicitIdxB =
                    oibvh_get_most_right_valid_implicitIdx(nextLevelB, leafLevB, virtualCountB);
                expandNodeCountA =
                    min((1 << (nextLevelA - levelA)), mostRightValidImplicitIdxA - startImplicitIdxA + 1);
                expandNodeCountB =
                    min((1 << (nextLevelB - levelB)), mostRightValidImplicitIdxB - startImplicitIdxB + 1);
            }
            numData = expandNodeCountA * expandNodeCountB;
        }
    }

    __shared__ bvtt_node_t bvttNodes[LOCALMEMSIZE];
    __shared__ unsigned int C;
    __shared__ unsigned int baseOffset;
    C = 0;
    int times = 0;
    int limitNumData = blockDim.x * (1 << (2 * expandLevels));
    unsigned int c;
    unsigned int checkpoint = 0;
    int indexOffsetA = 0;
    int indexOffsetB = 0;
    __syncthreads();

    do
    {
        if (numData > 0)
        {
            c = atomicAdd(&C, numData) - checkpoint;
            if (c < LOCALMEMSIZE)
            {
                unsigned int r = LOCALMEMSIZE - c; // remaining space
                unsigned int w = min(numData, r);  // amount written
                // write bvtt nodes to share memory
                for (int cnt = 0; cnt < w; cnt++)
                {
                    realIndexA = oibvh_implicit_to_real(startImplicitIdxA + indexOffsetA, leafLevA, virtualCountA);
                    realIndexB = oibvh_implicit_to_real(startImplicitIdxB + indexOffsetB, leafLevB, virtualCountB);
                    bvttNodes[c + cnt] = bvtt_node_t{realIndexA + aabbOffsetA, realIndexB + aabbOffsetB};
                    indexOffsetB++;
                    if (indexOffsetB % expandNodeCountB == 0)
                    {
                        indexOffsetB = 0;
                        indexOffsetA++;
                    }
                }
                numData = numData - w;
            }
        }
        __syncthreads();
        unsigned int s = min(C - checkpoint, LOCALMEMSIZE);
        if (s > 0)
        {
            checkpoint = C;
            if (localIndex == 0)
            {
                baseOffset = atomicAdd(nextBvttSize, s);
            }
            __syncthreads();
            // write bvtt node from share memory to global memory
            for (int i = 0; i < (LOCALMEMSIZE + blockDim.x - 1) / blockDim.x; i++)
            {
                if (i * blockDim.x + localIndex >= s)
                {
                    break;
                }
                dst[baseOffset + i * blockDim.x + localIndex] = bvttNodes[i * blockDim.x + localIndex];
            }
            __syncthreads();
        }
        else
        {
            break;
        }
        times++;
    } while (times * LOCALMEMSIZE < limitNumData);
}

namespace
{

// Projects the vertices of two triangles onto the given axis and checks for overlap.
// Returns true if the projections overlap (i.e., no separating axis found).
__device__ bool project6(const glm::vec3& ax,
                         const glm::vec3& p1,
                         const glm::vec3& p2,
                         const glm::vec3& p3,
                         const glm::vec3& q1,
                         const glm::vec3& q2,
                         const glm::vec3& q3)
{
    // Project triangle A vertices onto axis
    float P1 = dot(ax, p1);
    float P2 = dot(ax, p2);
    float P3 = dot(ax, p3);

    // Project triangle B vertices onto axis
    float Q1 = dot(ax, q1);
    float Q2 = dot(ax, q2);
    float Q3 = dot(ax, q3);

    // Find min/max projections for both triangles
    float mx1 = fmax(fmax(P1, P2), P3);
    float mn1 = fmin(fmin(P1, P2), P3);
    float mx2 = fmax(fmax(Q1, Q2), Q3);
    float mn2 = fmin(fmin(Q1, Q2), Q3);

    // Check for overlap in projections
    return (mn1 <= mx2) && (mn2 <= mx1);
}

} // namespace

__device__ bool triangles_intersect(const glm::vec3& P1,
                                    const glm::vec3& P2,
                                    const glm::vec3& P3,
                                    const glm::vec3& Q1,
                                    const glm::vec3& Q2,
                                    const glm::vec3& Q3)
{
    // One triangle is (p1,p2,p3).  Other is (q1,q2,q3).
    // Edges are (e1,e2,e3) and (f1,f2,f3).
    // Normals are n1 and m1
    // Outwards are (g1,g2,g3) and (h1,h2,h3).
    //
    // We assume that the triangle vertices are in the same coordinate system.
    //
    // First thing we do is establish a new c.s. so that p1 is at (0,0,0).

    glm::vec3 p1, p2, p3;
    glm::vec3 q1, q2, q3;
    glm::vec3 e1, e2, e3;
    glm::vec3 f1, f2, f3;
    glm::vec3 g1, g2, g3;
    glm::vec3 h1, h2, h3;
    glm::vec3 n1, m1;

    glm::vec3 ef11, ef12, ef13;
    glm::vec3 ef21, ef22, ef23;
    glm::vec3 ef31, ef32, ef33;

    p1 = glm::vec3(0, 0, 0);
    p2 = P2 - P1;
    p3 = P3 - P1;

    q1 = Q1 - P1;
    q2 = Q2 - P1;
    q3 = Q3 - P1;

    e1 = p2 - p1;
    e2 = p3 - p2;
    e3 = p1 - p3;

    f1 = q2 - q1;
    f2 = q3 - q2;
    f3 = q1 - q3;

    n1 = cross(e1, e2);
    m1 = cross(f1, f2);

    g1 = cross(e1, n1);
    g2 = cross(e2, n1);
    g3 = cross(e3, n1);
    h1 = cross(f1, m1);
    h2 = cross(f2, m1);
    h3 = cross(f3, m1);

    ef11 = cross(e1, f1);
    ef12 = cross(e1, f2);
    ef13 = cross(e1, f3);
    ef21 = cross(e2, f1);
    ef22 = cross(e2, f2);
    ef23 = cross(e2, f3);
    ef31 = cross(e3, f1);
    ef32 = cross(e3, f2);
    ef33 = cross(e3, f3);

    // now begin the series of tests

    return project6(n1, p1, p2, p3, q1, q2, q3) && project6(m1, p1, p2, p3, q1, q2, q3) &&
        project6(ef11, p1, p2, p3, q1, q2, q3) && project6(ef12, p1, p2, p3, q1, q2, q3) &&
        project6(ef13, p1, p2, p3, q1, q2, q3) && project6(ef21, p1, p2, p3, q1, q2, q3) &&
        project6(ef22, p1, p2, p3, q1, q2, q3) && project6(ef23, p1, p2, p3, q1, q2, q3) &&
        project6(ef31, p1, p2, p3, q1, q2, q3) && project6(ef32, p1, p2, p3, q1, q2, q3) &&
        project6(ef33, p1, p2, p3, q1, q2, q3) && project6(g1, p1, p2, p3, q1, q2, q3) &&
        project6(g2, p1, p2, p3, q1, q2, q3) && project6(g3, p1, p2, p3, q1, q2, q3) &&
        project6(h1, p1, p2, p3, q1, q2, q3) && project6(h2, p1, p2, p3, q1, q2, q3) &&
        project6(h3, p1, p2, p3, q1, q2, q3);
}

__global__ void triangle_intersect_kernel(tri_pair_node_t* triPairs,
                                          glm::uvec3* primitives,
                                          glm::vec3* vertices,
                                          unsigned int* primOffsets,
                                          unsigned int* vertexOffsets,
                                          int_tri_pair_node_t* intTriPairs,
                                          unsigned int* intTriPairCount,
                                          unsigned int layoutLength,
                                          unsigned int triPairCount)
{
    unsigned int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int localIndex = threadIdx.x;
    __shared__ unsigned int sharedPrimOffsets[256];
    __shared__ unsigned int sharedVertexOffsets[256];
    if (localIndex < layoutLength)
    {
        sharedPrimOffsets[localIndex] = primOffsets[localIndex];
        sharedVertexOffsets[localIndex] = vertexOffsets[localIndex];
    }
    __syncthreads();
    if (globalIndex >= triPairCount)
        return;

    tri_pair_node_t node = triPairs[globalIndex];
    unsigned int triIndexA = node.m_triIndex[0];
    unsigned int triIndexB = node.m_triIndex[1];
    unsigned int bvhIndexA, bvhIndexB, primOffsetA, primOffsetB, vertexOffsetA, vertexOffsetB;
    read_information(
        sharedPrimOffsets, sharedVertexOffsets, triIndexA, layoutLength, bvhIndexA, primOffsetA, vertexOffsetA);
    read_information(
        sharedPrimOffsets, sharedVertexOffsets, triIndexB, layoutLength, bvhIndexB, primOffsetB, vertexOffsetB);
    glm::uvec3 triangleA = primitives[triIndexA];
    glm::uvec3 triangleB = primitives[triIndexB];

    glm::vec3 triVerticesA[3], triVerticesB[3];
    for (int i = 0; i < 3; i++)
    {
        triVerticesA[i] = vertices[vertexOffsetA + triangleA[i]];
        triVerticesB[i] = vertices[vertexOffsetB + triangleB[i]];
    }

    // triangle intersect
    if (triangles_intersect(
            triVerticesA[0], triVerticesA[1], triVerticesA[2], triVerticesB[0], triVerticesB[1], triVerticesB[2]))
    {
        unsigned int intTriPairOffset = atomicAdd(intTriPairCount, 1u);
        int_tri_pair_node_t intTriPair{bvhIndexA, bvhIndexB, triIndexA - primOffsetA, triIndexB - primOffsetB};
        intTriPairs[intTriPairOffset] = intTriPair;
    }
}